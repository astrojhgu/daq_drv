#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <complex>
#include <vector>
#include <hip/hip_runtime.h>
#include "errchk.h"
#include <cstdint>
constexpr int BINSIZE_I=4;
constexpr int BINSIZE_J=4;
constexpr int MAX_BLOCK_X=1024;



__global__ void correlate_gpu(float* pdataIn,float* pdataOut,int nch,int nnodes,int ntime, int ch_seg, int length_per_node)
{
  int i=blockIdx.x;
  int j=blockIdx.y;
  
  if(j<i)
  {
    return;
  }
  int ch_seg_id=blockIdx.z;
  int ch_shift=ch_seg*ch_seg_id;
  int ch_id=ch_shift+threadIdx.x;

  if(ch_id>=nch)
  {
    return;
  }



  float cr=0;
  float ci=0;
  float r1,r2,i1,i2;
  cr=0;
  ci=0;
  
  for(int m=0;m<ntime;++m)
    {
       r1 = pdataIn[i*length_per_node+m*nch*2+2*ch_id];
       i1 = pdataIn[i*length_per_node+m*nch*2+2*ch_id+1];
       r2 = pdataIn[j*length_per_node+m*nch*2+2*ch_id];
       i2 = pdataIn[j*length_per_node+m*nch*2+2*ch_id+1];

      cr+=r1*r2+i1*i2;
      if(i!=j)
	    {
	      ci+=i1*r2-r1*i2;
	    }
    }

  int bl=(2*nnodes-1-i)*i/2+j;
  pdataOut[bl*2*nch+2*ch_id]+=cr/ntime;
  pdataOut[bl*2*nch+2*ch_id+1]+=ci/ntime;
}

extern "C" void correlate_c(std::complex<float>* input_data,std::complex<float>* output_data,int nch,int nnodes,int ntime)
{
  int by=(nch-1)/MAX_BLOCK_X+1;
  int ch_seg=nch/by;
  dim3 gd(nnodes,nnodes,by);
  int bx=std::min(nch, MAX_BLOCK_X);
  //std::cout<<bx<<" "<<by<<std::endl;
  dim3 bd(bx,1,1);
  int length_per_node=nch*ntime;

  correlate_gpu<<<gd, bd>>>((float*)input_data, (float*)output_data, nch,nnodes,ntime, ch_seg, length_per_node); 

}

#if 0

int main(int argc, char* argv[]){
    hipInit(0);
    hipDeviceReset();

    int nch=8192;
    int ntime=65536/2;
    int nnodes=2;

    
    int nbaselines=nnodes*(nnodes+1)/2;


    int input_length=nch*ntime*nnodes;
    int output_length=nbaselines*nch;

    std::cout<<input_length<<std::endl;

    std::complex<float>* host_input_data=nullptr;
    std::complex<float>* host_output_data=nullptr;

    gpuErrchk(hipHostMalloc((void**)&host_input_data, input_length*sizeof(float)*2));
    gpuErrchk(hipHostMalloc((void**)&host_output_data, output_length*sizeof(float)*2));


    for(int i=0;i<input_length;++i){
        host_input_data[i]=std::complex<float>(1,1);
    }

    std::complex<float>* input_data;
    std::complex<float>* output_data;

    gpuErrchk(hipMalloc((void**)&input_data, input_length*sizeof(float)*2));
    gpuErrchk(hipMalloc((void**)&output_data, output_length*sizeof(float)*2));

    
    for(int i=0;i<100;++i){
        std::cout<<i<<std::endl;
        std::cout<<"beg"<<std::endl;
        gpuErrchk(hipMemcpy((void*)input_data, (void*)host_input_data, input_length*sizeof(float)*2, hipMemcpyHostToDevice));

        hipMemset(output_data, 0, output_length*sizeof(float)*2);
        correlate_c(input_data,output_data,nch,nnodes,ntime);

        gpuErrchk(hipMemcpy((void*)host_output_data, (void*)output_data, output_length*sizeof(float)*2, hipMemcpyDeviceToHost));
        std::cout<<host_output_data[(nbaselines-1)*nch+nch-1]<<std::endl;
        gpuErrchk(hipDeviceSynchronize());          
        
    }

    std::cout<<"Hello"<<std::endl;
}

#endif